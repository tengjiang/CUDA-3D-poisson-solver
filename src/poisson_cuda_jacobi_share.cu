/*
  Created and Maintained by Teng Jiang @ Peking University.
  Date: 30/11/2021
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#define checkCudaErrors( a ) do { \
if (hipSuccess != (a)) { \
fprintf(stderr, "Cuda runtime error in line %d of file %s \
: %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
exit(EXIT_FAILURE); \
} \
} while(0);


#define N 512
#define MAXITER 100
#define RTOL 1e-6
#define PI 3.14159265358979323846


void init_sol_h(double *__restrict__ b, double *__restrict__ u_exact, double *__restrict__ u)
{
    double a = N / 4.;
    double h = 1. / (N + 1);
#pragma omp parallel for collapse(3)
    for (int i = 0; i < N + 2; i++)
        for (int j = 0; j < N + 2; j++)
            for (int k = 0; k < N + 2; k++)
            {
                u_exact[i * (N + 2) * (N + 2) + j * (N + 2) + k] = sin(a * PI * i * h) * sin(a * PI * j * h) * sin(a * PI * k * h);
                u[i * (N + 2) * (N + 2) + j * (N + 2) + k] = 0.;
            }
            
#pragma omp parallel for collapse(3)
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            for (int k = 0; k < N; k++)
            {
                b[i * N * N + j * N + k] = 3. * a * a * PI * PI * sin(a * PI * (i + 1) * h) * sin(a * PI * (j + 1) * h) * sin(a * PI * (k + 1) * h) * h * h;
            }
}

double error(double *__restrict__ u, double *__restrict__ u_exact)
{
    double tmp = 0;
#pragma omp parallel for reduction(+:tmp)
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            for (int k = 0; k < N; k++)
            {
                tmp += pow((u_exact[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1] - u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1]), 2);
            }
    double tmp2 = 0;
#pragma omp parallel for reduction(+:tmp2)
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            for (int k = 0; k < N; k++)
            {
                tmp2 += pow((u_exact[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1]), 2);
            }
    return pow(tmp, 0.5) / pow(tmp2, 0.5);
}

__global__
void residual_norm_kernel(double *__restrict__ d_u, double *__restrict__ b,double *__restrict__ rn){
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int i = blockDim.z * blockIdx.z + threadIdx.z;
    __shared__ double u[2+4][8+2][32+2];

    u[threadIdx.z+1][threadIdx.y+1][threadIdx.x+1]= d_u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1];
    if(threadIdx.x == 0) u[threadIdx.z+1][threadIdx.y+1][threadIdx.x]= d_u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 0];
    if(threadIdx.x == blockDim.x-1) u[threadIdx.z+1][threadIdx.y+1][threadIdx.x+2]= d_u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 2];
    if(threadIdx.y == 0) u[threadIdx.z+1][threadIdx.y][threadIdx.x+1]= d_u[(i + 1) * (N + 2) * (N + 2) + (j + 0) * (N + 2) + k + 1];
    if(threadIdx.y == blockDim.y-1) u[threadIdx.z+1][threadIdx.y+2][threadIdx.x+1]= d_u[(i + 1) * (N + 2) * (N + 2) + (j + 2) * (N + 2) + k + 1];
    if(threadIdx.z == 0) u[threadIdx.z][threadIdx.y+1][threadIdx.x+1]= d_u[(i + 0) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1];
    if(threadIdx.z == blockDim.z-1) u[threadIdx.z+2][threadIdx.y+1][threadIdx.x+1]= d_u[(i + 2) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1];

    __syncthreads();

    double r = b[i * N * N + j * N + k] + 
                    + u[threadIdx.z+1][threadIdx.y+2][threadIdx.x+1]
                                + u[threadIdx.z+1][threadIdx.y][threadIdx.x+1]
                                + u[threadIdx.z+1][threadIdx.y+1][threadIdx.x]
                                + u[threadIdx.z+1][threadIdx.y+1][threadIdx.x+2]
                                + u[threadIdx.z][threadIdx.y+1][threadIdx.x+1]
                                + u[threadIdx.z+2][threadIdx.y+1][threadIdx.x+1]
                    - 6.0 * u[threadIdx.z+1][threadIdx.y+1][threadIdx.x+1];
    rn[i * N * N + j * N + k]=r*r;
    __syncthreads();
}

__global__
void jacobi_kernel(double *__restrict__ d_u, double *__restrict__ d_u_new, double *__restrict__ d_b){
    __shared__ double u[2+4][8+2][32+2];
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int i = blockDim.z * blockIdx.z + threadIdx.z;
    u[threadIdx.z+1][threadIdx.y+1][threadIdx.x+1]= d_u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1];
    if(threadIdx.x == 0) u[threadIdx.z+1][threadIdx.y+1][threadIdx.x]= d_u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 0];
    if(threadIdx.x == blockDim.x-1) u[threadIdx.z+1][threadIdx.y+1][threadIdx.x+2]= d_u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 2];
    if(threadIdx.y == 0) u[threadIdx.z+1][threadIdx.y][threadIdx.x+1]= d_u[(i + 1) * (N + 2) * (N + 2) + (j + 0) * (N + 2) + k + 1];
    if(threadIdx.y == blockDim.y-1) u[threadIdx.z+1][threadIdx.y+2][threadIdx.x+1]= d_u[(i + 1) * (N + 2) * (N + 2) + (j + 2) * (N + 2) + k + 1];
    if(threadIdx.z == 0) u[threadIdx.z][threadIdx.y+1][threadIdx.x+1]= d_u[(i + 0) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1];
    if(threadIdx.z == blockDim.z-1) u[threadIdx.z+2][threadIdx.y+1][threadIdx.x+1]= d_u[(i + 2) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1];

    __syncthreads();

    d_u_new[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1] = 
                                (d_b[i * N * N + j * N + k] 
                                + u[threadIdx.z+1][threadIdx.y+2][threadIdx.x+1]
                                + u[threadIdx.z+1][threadIdx.y][threadIdx.x+1]
                                + u[threadIdx.z+1][threadIdx.y+1][threadIdx.x]
                                + u[threadIdx.z+1][threadIdx.y+1][threadIdx.x+2]
                                + u[threadIdx.z][threadIdx.y+1][threadIdx.x+1]
                                + u[threadIdx.z+2][threadIdx.y+1][threadIdx.x+1]
                                ) / 6.0;
    __syncthreads();
}

double residual_norm(double *__restrict__ u, double *__restrict__ b)
{
    double norm2 = 0;

    #pragma omp parallel for collapse(3) reduction(+:norm2)
    for (int i = 0; i < N; i ++)
    {
        for (int j = 0; j < N; j ++)
        {
            for (int k = 0; k < N; k++)
            {
                double r = b[i * N * N + j * N + k] + 
                    + u[(i + 0) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1]
                    + u[(i + 1) * (N + 2) * (N + 2) + (j + 0) * (N + 2) + k + 1]
                    + u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 0] 
                    + u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 2]
                    + u[(i + 1) * (N + 2) * (N + 2) + (j + 2) * (N + 2) + k + 1]
                    + u[(i + 2) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1]
                    - 6.0 * u[(i + 1) * ((N + 2) * (N + 2)) + (j + 1) * (N + 2) + (k + 1)];
                norm2 += r * r;
            }
        }
    }
    return sqrt(norm2);
}

__global__ void reduction(double *g_odata, double *g_idata)
{
    // dynamically allocated shared memory

    __shared__  double temp[N];

    int tid = threadIdx.x;

    // first, each thread loads data into shared memory

    temp[tid] = g_idata[tid+blockIdx.x*blockDim.x];

    // next, we perform binary tree reduction

    for (int d = blockDim.x>>1; d > 0; d >>= 1) {
      __syncthreads();  // ensure previous step completed 
      if (tid<d)  temp[tid] += temp[tid+d];
    }

    // finally, first thread puts result into global memory

    if (tid==0) g_odata[blockIdx.x] = temp[0];

}

template <unsigned int blockSize>
__device__ void warpReduce(volatile double *sdata, unsigned int tid) {
if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}
template <unsigned int blockSize>
__global__ void reduce6(double *g_idata, double *g_odata) {
__shared__ double sdata[N];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
__syncthreads();
if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
if (tid < 32) warpReduce<blockSize>(sdata, tid);
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main(int argc, char **argv)
{
    double * u = (double *)malloc(sizeof(double) * (N + 2) * (N + 2) * (N + 2));
    double * u_exact = (double *)malloc(sizeof(double) * (N + 2) * (N + 2) * (N + 2));
    double * b = (double *)malloc(sizeof(double) * N * N * N);
    double * temp; //used to swap pointer
    double * h_recv = (double *)malloc(sizeof(double));

    init_sol_h(b, u_exact, u);
    double normr0 = residual_norm(u, b); // Please ensure that this residual_norm is exact.
    double normr = normr0;

    printf("Method: Jacobi with shared memory\n");
    
    // Allocate device memory and transfer data
    double  *d_u_exact, *d_u, *d_b,*d_u_new, *d_rn, *d_rn_out; // d_u_new for jacobi, d_rn for residual norms, d_rn_out for results after reduction
    hipMalloc((void**)&d_u,sizeof(double)* (N + 2) * (N + 2) * (N + 2));
    hipMalloc((void**)&d_u_new,sizeof(double)* (N + 2) * (N + 2) * (N + 2));
    hipMalloc((void**)&d_b,sizeof(double)*N * N * N);
    hipMalloc((void**)&d_u_exact,sizeof(double)* (N + 2) * (N + 2) * (N + 2));
    hipMalloc((void**)&d_rn,sizeof(double)*  N*N*N);
    hipMalloc((void**)&d_rn_out,sizeof(double)*  N*N*N);

    printf("cuda malloc successful\n");
    hipMemset(d_rn,0,sizeof(double)* N*N*N); 
    hipMemset(d_rn_out,0,sizeof(double)*  N*N*N); 
    hipMemset(d_u,0,sizeof(double)* (N + 2) * (N + 2) * (N + 2)); 
    hipMemset(d_u_new,0,sizeof(double)* (N + 2) * (N + 2) * (N + 2)); 
    hipMemcpy(d_u_exact, u_exact,sizeof(double)* (N + 2) * (N + 2) * (N + 2), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b,sizeof(double)*N* N* N, hipMemcpyHostToDevice);
    /*
    checkCudaErrors(cudaMemset(d_rn,0,sizeof(double)* N*N*N)); 
    checkCudaErrors(cudaMemset(d_rn_out,0,sizeof(double)*  N*N*N)); 
    checkCudaErrors(cudaMemset(d_u,0,sizeof(double)* (N + 2) * (N + 2) * (N + 2))); 
    checkCudaErrors(cudaMemset(d_u_new,0,sizeof(double)* (N + 2) * (N + 2) * (N + 2))); 
    checkCudaErrors(cudaMemcpy(d_u_exact, u_exact,sizeof(double)* (N + 2) * (N + 2) * (N + 2), cudaMemcpyHostToDevice));
    checkCudaErrors(cudaMemcpy(d_b, b,sizeof(double)*N* N* N, cudaMemcpyHostToDevice));
    */

    printf("cuda copy and set successful\n");

    dim3 grid_dim(N/32, N/8, N/4);
    dim3 block_dim(32,8,4);
    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop);
    
    
    int tsteps = MAXITER;
    hipEventRecord(start, 0);
    for (int k = 0; k < MAXITER; k++)
    {   
        printf("Iteration %d, normr/normr0=%g\n", k,  normr/ normr0);

        jacobi_kernel<<<grid_dim, block_dim>>>(d_u,d_u_new, d_b);
        //cudaDeviceSynchronize();

        //switch pointer
        temp = d_u_new;
        d_u_new = d_u;
        d_u = temp;
        
        residual_norm_kernel<<<grid_dim, block_dim>>>(d_u, d_b, d_rn);

        
        //checkCudaErrors(cudaDeviceSynchronize());
        reduction<<<N*N,N>>>(d_rn_out,d_rn);
        //checkCudaErrors(cudaDeviceSynchronize());
        reduction<<<N,N>>>(d_rn,d_rn_out);
        //checkCudaErrors(cudaDeviceSynchronize());
        reduction<<<1,N>>>(d_rn_out,d_rn);
        
        /*
        reduce6<N><<< N*N, N>>>(d_rn, d_rn_out);
        reduce6<N><<< N, N>>>(d_rn_out,d_rn);
        reduce6<N><<< 1, N>>>(d_rn, d_rn_out);
        */
        //cudaDeviceSynchronize();
        
        hipMemcpy(h_recv, d_rn_out ,sizeof(double), hipMemcpyDeviceToHost);
        
        normr = sqrt(h_recv[0]);
        if (normr < RTOL * normr0)
        {
            printf("Iteration %d, normr/normr0=%g\n", k + 1, normr/ normr0);
            tsteps = k + 1;
            printf("Converged with %d iterations.\n", tsteps);
            break;
        }
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime; 
    hipEventElapsedTime(&elapsedTime, start, stop);
    //printf("time: %g\n", elapsedTime);
    printf("time: %g\n", elapsedTime / 1000.);

    hipMemcpy(u, d_u ,sizeof(double)* (N + 2) * (N + 2) * (N + 2), hipMemcpyDeviceToHost);
    printf("  grid  dim:  %d, %d, %d.\n", grid_dim.x, grid_dim.y, grid_dim.z);
    printf("  block dim: %d, %d, %d.\n", block_dim.x, block_dim.y, block_dim.z);
    hipFree(d_u_exact); hipFree(d_u); hipFree(d_b); hipFree(d_rn); hipFree(d_rn_out); 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Residual norm: %g\n", normr);

    double final_normr = residual_norm(u, b); // Please ensure that this residual_norm is exact.
    printf("Final residual norm: %g\n", final_normr);
    printf("|r_n|/|r_0| = %g\n", final_normr / normr0);

    long long residual_norm_bytes = sizeof(double) * ((N + 2) * (N + 2) * (N + 2) + (N * N * N)) * tsteps;
    long long gs_bytes = sizeof(double) * ((N + 2) * (N + 2) * (N + 2) + 2 * (N * N * N)) * tsteps;

    long long total_bytes = residual_norm_bytes + gs_bytes;
    double bandwidth = total_bytes / elapsedTime * 1000.;

    printf("total bandwidth: %g GB/s\n", bandwidth / (double)(1 << 30));

    double relative_err = error(u, u_exact);
    printf("relative error: %g\n", relative_err);
    
    //int num_threads = omp_get_max_threads();
    //printf("openmp max num threads: %d\n", num_threads);

    free(u);
    free(u_exact);
    free(b);

    return 0;
}
