/*
  Created and Maintained by Teng Jiang @ Peking University.
  Date: 30/11/2021
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#define checkCudaErrors( a ) do { \
if (hipSuccess != (a)) { \
fprintf(stderr, "Cuda runtime error in line %d of file %s \
: %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
exit(EXIT_FAILURE); \
} \
} while(0);


#define N 512
#define MAXITER 100
#define MINITER 34
#define RTOL 1e-6
#define PI 3.14159265358979323846


void init_sol_h(double *__restrict__ b, double *__restrict__ u_exact, double *__restrict__ u)
{
    double a = N / 4.;
    double h = 1. / (N + 1);
#pragma omp parallel for
    for (int i = 0; i < N + 2; i++)
        for (int j = 0; j < N + 2; j++)
            for (int k = 0; k < N + 2; k++)
            {
                u_exact[i * (N + 2) * (N + 2) + j * (N + 2) + k] = sin(a * PI * i * h) * sin(a * PI * j * h) * sin(a * PI * k * h);
                u[i * (N + 2) * (N + 2) + j * (N + 2) + k] = 0.;
            }
            
#pragma omp parallel for
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            for (int k = 0; k < N; k++)
            {
                b[i * N * N + j * N + k] = 3. * a * a * PI * PI * sin(a * PI * (i + 1) * h) * sin(a * PI * (j + 1) * h) * sin(a * PI * (k + 1) * h) * h * h;
            }
}

double error(double *__restrict__ u, double *__restrict__ u_exact)
{
    double tmp = 0;
#pragma omp parallel for reduction(+:tmp)
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            for (int k = 0; k < N; k++)
            {
                tmp += pow((u_exact[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1] - u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1]), 2);
            }
    double tmp2 = 0;
#pragma omp parallel for reduction(+:tmp2)
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            for (int k = 0; k < N; k++)
            {
                tmp2 += pow((u_exact[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1]), 2);
            }
    return pow(tmp, 0.5) / pow(tmp2, 0.5);
}

__global__
void gauss_seidel_kernel(double *__restrict__ d_u, double *__restrict__ d_b,int num){
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int i = blockDim.z * blockIdx.z + threadIdx.z;
   
    if((i+j+k+num)%2==0){ // 判断颜色
    d_u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1] = 
                                (d_b[i * N * N + j * N + k] 
                                + d_u[(i + 0) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1]  // from last iter
                                + d_u[(i + 1) * (N + 2) * (N + 2) + (j + 0) * (N + 2) + k + 1]
                                + d_u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 0] // from last iter
                                + d_u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 2]
                                + d_u[(i + 1) * (N + 2) * (N + 2) + (j + 2) * (N + 2) + k + 1]  // from last iter
                                + d_u[(i + 2) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1]
                                ) / 6.0;}
    //__syncthreads();
}



__global__
void residual_norm_kernel(double *__restrict__ u, double *__restrict__ b,double *__restrict__ rn){
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int i = blockDim.z * blockIdx.z + threadIdx.z;
    double r = b[i * N * N + j * N + k] + 
                    + u[(i + 0) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1]
                    + u[(i + 1) * (N + 2) * (N + 2) + (j + 0) * (N + 2) + k + 1]
                    + u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 0] 
                    + u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 2]
                    + u[(i + 1) * (N + 2) * (N + 2) + (j + 2) * (N + 2) + k + 1]
                    + u[(i + 2) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1]
                    - 6.0 * u[(i + 1) * ((N + 2) * (N + 2)) + (j + 1) * (N + 2) + (k + 1)];
    rn[i * N * N + j * N + k]=r*r;
    __syncthreads();
}

double residual_norm(double *__restrict__ u, double *__restrict__ b)
{
    double norm2 = 0;

    #pragma omp parallel for collapse(3) reduction(+:norm2)
    for (int i = 0; i < N; i ++)
    {
        for (int j = 0; j < N; j ++)
        {
            for (int k = 0; k < N; k++)
            {
                double r = b[i * N * N + j * N + k] + 
                    + u[(i + 0) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1]
                    + u[(i + 1) * (N + 2) * (N + 2) + (j + 0) * (N + 2) + k + 1]
                    + u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 0] 
                    + u[(i + 1) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 2]
                    + u[(i + 1) * (N + 2) * (N + 2) + (j + 2) * (N + 2) + k + 1]
                    + u[(i + 2) * (N + 2) * (N + 2) + (j + 1) * (N + 2) + k + 1]
                    - 6.0 * u[(i + 1) * ((N + 2) * (N + 2)) + (j + 1) * (N + 2) + (k + 1)];
                norm2 += r * r;
            }
        }
    }
    return sqrt(norm2);
}

__global__ void reduction(double *g_odata, double *g_idata)
{
    // dynamically allocated shared memory

    __shared__  double temp[N];

    int tid = threadIdx.x;

    // first, each thread loads data into shared memory

    temp[tid] = g_idata[tid+blockIdx.x*blockDim.x];

    // next, we perform binary tree reduction

    for (int d = blockDim.x>>1; d > 0; d >>= 1) {
      __syncthreads();  // ensure previous step completed 
      if (tid<d)  temp[tid] += temp[tid+d];
    }

    // finally, first thread puts result into global memory

    if (tid==0) g_odata[blockIdx.x] = temp[0];
}

int main(int argc, char **argv)
{
    double * u = (double *)malloc(sizeof(double) * (N + 2) * (N + 2) * (N + 2));
    double * u_exact = (double *)malloc(sizeof(double) * (N + 2) * (N + 2) * (N + 2));
    double * b = (double *)malloc(sizeof(double) * N * N * N);
    double * h_recv = (double *)malloc(sizeof(double));

    init_sol_h(b, u_exact, u);
    double normr0 = residual_norm(u, b); // Please ensure that this residual_norm is exact.
    double normr = normr0;

    printf("Method: Gauss Seidel with red and black coloring\n");
    
    // Allocate device memory and transfer data
    double  *d_u_exact, *d_u, *d_b,*d_u_new, *d_rn, *d_rn_out; // d_u_new for jacobi, d_rn for residual norms, d_rn_out for results after reduction
    hipMalloc((void**)&d_u,sizeof(double)* (N + 2) * (N + 2) * (N + 2));
    hipMalloc((void**)&d_u_new,sizeof(double)* (N + 2) * (N + 2) * (N + 2));
    hipMalloc((void**)&d_b,sizeof(double)*N * N * N);
    hipMalloc((void**)&d_u_exact,sizeof(double)* (N + 2) * (N + 2) * (N + 2));
    hipMalloc((void**)&d_rn,sizeof(double)*  N*N*N);
    hipMalloc((void**)&d_rn_out,sizeof(double)*  N*N*N);

    printf("cuda malloc successful\n");
    hipMemset(d_rn,0,sizeof(double)* N*N*N); 
    hipMemset(d_rn_out,0,sizeof(double)*  N*N*N); 
    hipMemset(d_u,0,sizeof(double)* (N + 2) * (N + 2) * (N + 2)); 
    hipMemset(d_u_new,0,sizeof(double)* (N + 2) * (N + 2) * (N + 2)); 
    hipMemcpy(d_u_exact, u_exact,sizeof(double)* (N + 2) * (N + 2) * (N + 2), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b,sizeof(double)*N* N* N, hipMemcpyHostToDevice);
    /*
    checkCudaErrors(cudaMemset(d_rn,0,sizeof(double)* N*N*N)); 
    checkCudaErrors(cudaMemset(d_rn_out,0,sizeof(double)*  N*N*N)); 
    checkCudaErrors(cudaMemset(d_u,0,sizeof(double)* (N + 2) * (N + 2) * (N + 2))); 
    checkCudaErrors(cudaMemset(d_u_new,0,sizeof(double)* (N + 2) * (N + 2) * (N + 2))); 
    checkCudaErrors(cudaMemcpy(d_u_exact, u_exact,sizeof(double)* (N + 2) * (N + 2) * (N + 2), cudaMemcpyHostToDevice));
    checkCudaErrors(cudaMemcpy(d_b, b,sizeof(double)*N* N* N, cudaMemcpyHostToDevice));
    */

    printf("cuda copy and set successful\n");

    dim3 grid_dim(N/32, N/8, N/4);
    dim3 block_dim(32,8,4);
    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop);
    
    
    int tsteps = MAXITER;
    hipEventRecord(start, 0);
    for (int k = 0; k < MAXITER; k++)
    {   
        printf("Iteration %d, normr/normr0=%g\n", k,  normr/ normr0);

        gauss_seidel_kernel<<<grid_dim, block_dim>>>(d_u, d_b,1);
        gauss_seidel_kernel<<<grid_dim, block_dim>>>(d_u, d_b,0);

        residual_norm_kernel<<<grid_dim, block_dim>>>(d_u, d_b, d_rn);
        //checkCudaErrors(cudaDeviceSynchronize());
        reduction<<<N*N,N>>>(d_rn_out,d_rn);
        //checkCudaErrors(cudaDeviceSynchronize());
        reduction<<<N,N>>>(d_rn,d_rn_out);
        //checkCudaErrors(cudaDeviceSynchronize());
        reduction<<<1,N>>>(d_rn_out,d_rn);
        
        hipMemcpy(h_recv, d_rn_out ,sizeof(double), hipMemcpyDeviceToHost);
        
        normr = sqrt(h_recv[0]);
        if (normr < RTOL * normr0 && k+1>=MINITER)
        {
            printf("Iteration %d, normr/normr0=%g\n", k + 1, normr/ normr0);
            tsteps = k + 1;
            printf("Converged with %d iterations.\n", tsteps);
            break;
        }
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime; 
    hipEventElapsedTime(&elapsedTime, start, stop);
    //printf("time: %g\n", elapsedTime);
    printf("time: %g\n", elapsedTime / 1000.);

    hipMemcpy(u, d_u ,sizeof(double)* (N + 2) * (N + 2) * (N + 2), hipMemcpyDeviceToHost);
    printf("  grid  dim:  %d, %d, %d.\n", grid_dim.x, grid_dim.y, grid_dim.z);
    printf("  block dim: %d, %d, %d.\n", block_dim.x, block_dim.y, block_dim.z);
    hipFree(d_u_exact); hipFree(d_u); hipFree(d_b); hipFree(d_rn); hipFree(d_rn_out); 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Residual norm: %g\n", normr);

    double final_normr = residual_norm(u, b); // Please ensure that this residual_norm is exact.
    printf("Final residual norm: %g\n", final_normr);
    printf("|r_n|/|r_0| = %g\n", final_normr / normr0);

    long long residual_norm_bytes = sizeof(double) * ((N + 2) * (N + 2) * (N + 2) + (N * N * N)) * tsteps;
    long long gs_bytes = sizeof(double) * ((N + 2) * (N + 2) * (N + 2) + 2 * (N * N * N)) * tsteps;

    long long total_bytes = residual_norm_bytes + gs_bytes;
    double bandwidth = total_bytes / elapsedTime * 1000.;

    printf("total bandwidth: %g GB/s\n", bandwidth / (double)(1 << 30));

    double relative_err = error(u, u_exact);
    printf("relative error: %g\n", relative_err);
    
    //int num_threads = omp_get_max_threads();
    //printf("openmp max num threads: %d\n", num_threads);

    free(u);
    free(u_exact);
    free(b);

    return 0;
}
